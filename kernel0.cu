#include "hip/hip_runtime.h"

#include <assert.h>

#include "common.h"
#include "timer.h"

__global__ void nw_0(int* matrix, unsigned char* sequence1_d, unsigned char* sequence2_d, int* scores_d) {

	unsigned long long int base = SEQUENCE_LENGTH*SEQUENCE_LENGTH*blockIdx.x;
	unsigned int segment = SEQUENCE_LENGTH*blockIdx.x;
    unsigned int tidx = threadIdx.x;
	int row, col, top, left, topleft, insertion, deletion, match, max;

	for(unsigned int i = 0; i < SEQUENCE_LENGTH; ++i)
	{ 
		if(tidx <= i)
		{	

			row = i-tidx;
			col = tidx;
			
			top = (col == 0 && row==0)?DELETION:(col==0)?matrix[base + SEQUENCE_LENGTH*(row-1) + col]:(row==0)?(col+1)*DELETION:matrix[base + SEQUENCE_LENGTH*(row-1) + col];
			left = (col == 0 && row==0)?INSERTION:(col==0)?(row+1)*INSERTION:(row==0)?matrix[base + SEQUENCE_LENGTH*row + (col-1)]:matrix[base + SEQUENCE_LENGTH*row + (col-1)];
			topleft = (col == 0 && row==0)?0:(col==0)?row*INSERTION:(row==0)?col*DELETION:matrix[base + SEQUENCE_LENGTH*(row-1) + (col-1)];
			
			
			insertion = top + INSERTION;
			deletion = left + DELETION;
			match = topleft;

			if(sequence1_d[segment + col] == sequence2_d[segment + row]) {
				match += MATCH;
			}
			else {
				match += MISMATCH;
			}

			if(insertion > deletion) {
				max = insertion;
			}
			else {
				max = deletion;
			}

			if(match > max) {
				max = match;
			}

			matrix[base + SEQUENCE_LENGTH*row + col] = max;
		}				
		__syncthreads();
	}
        
	for(int i=SEQUENCE_LENGTH-1; i>0; --i)
	{
		if(tidx < i)
		{
			row = SEQUENCE_LENGTH - tidx - 1;
			col = SEQUENCE_LENGTH + tidx - i;

			top  = matrix[base + SEQUENCE_LENGTH*(row-1) + col];
			left = matrix[base + SEQUENCE_LENGTH*row + (col-1)];
			topleft = matrix[base + SEQUENCE_LENGTH*(row-1) + (col-1)];

			insertion = top + INSERTION;
			deletion = left + DELETION;
			match = topleft;

			if(sequence1_d[segment + col] == sequence2_d[segment + row]) {
				match += MATCH;
			}
			else {
				match += MISMATCH;
			}

			if(insertion > deletion) {
				max = insertion;
			}
			else {
				max = deletion;
			}

			if(match > max){
				max = match;
			}

			matrix[base + SEQUENCE_LENGTH*row + col] = max;
		}
		__syncthreads();
	}

	if(tidx == 0)
	{
		scores_d[blockIdx.x] = matrix[base + SEQUENCE_LENGTH*(SEQUENCE_LENGTH-1) + (SEQUENCE_LENGTH-1)];
	}
}


void nw_gpu0(unsigned char* sequence1_d, unsigned char* sequence2_d, int* scores_d, unsigned int numSequences) {
    
    assert(SEQUENCE_LENGTH <= 1024); // You can assume the sequence length is not more than 1024

	int* matrix;
    hipMalloc((void**)&matrix, sizeof(int)*SEQUENCE_LENGTH*SEQUENCE_LENGTH*numSequences);
	
	int numThreadsPerBlock = SEQUENCE_LENGTH;
    int numBlocks = numSequences;
	nw_0 <<<numBlocks, numThreadsPerBlock>>> (matrix, sequence1_d, sequence2_d, scores_d);

    hipFree(matrix);
}
