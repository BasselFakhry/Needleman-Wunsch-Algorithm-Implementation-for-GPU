#include "hip/hip_runtime.h"

#include <assert.h>

#include "common.h"
#include "timer.h"

__global__ void nw_3(unsigned char* sequence1_d, unsigned char* sequence2_d, int* scores_d) {

    

}

void nw_gpu3(unsigned char* sequence1_d, unsigned char* sequence2_d, int* scores_d, unsigned int numSequences) {

    assert(SEQUENCE_LENGTH <= 1024); // You can assume the sequence length is not more than 1024

    unsigned int numThreadsPerBlock = (SEQUENCE_LENGTH+COARSE_FACTOR-1)/COARSE_FACTOR;
    unsigned int numBlocks = numSequences;
    nw_3 <<<numBlocks, numThreadsPerBlock>>> (sequence1_d,sequence2_d,scores_d);
}
